#include "hip/hip_runtime.h"
#include "cupoch/geometry/voxelgrid.h"

#include "cupoch/camera/pinhole_camera_parameters.h"
#include "cupoch/geometry/boundingvolume.h"
#include "cupoch/geometry/image.h"
#include <thrust/iterator/discard_iterator.h>

using namespace cupoch;
using namespace cupoch::geometry;

namespace {

struct elementwise_min_functor {
    __device__
    Eigen::Vector3i operator()(const Eigen::Vector3i& a, const Eigen::Vector3i& b) {
        return a.array().min(b.array()).matrix();
    }
};

struct elementwise_max_functor {
    __device__
    Eigen::Vector3i operator()(const Eigen::Vector3i& a, const Eigen::Vector3i& b) {
        return a.array().max(b.array()).matrix();
    }
};

struct compute_center_functor {
    compute_center_functor(float voxel_size, const Eigen::Vector3f& origin, const Eigen::Vector3f& half_voxel_size)
     : voxel_size_(voxel_size), origin_(origin), half_voxel_size_(half_voxel_size) {};
    const float voxel_size_;
    const Eigen::Vector3f origin_;
    const Eigen::Vector3f half_voxel_size_;
    __device__
    Eigen::Vector3f operator()(const Eigen::Vector3i& x) const {
        return x.cast<float>() * voxel_size_ + origin_ + half_voxel_size_;
    }
};

struct extract_grid_index_functor {
    __device__
    Eigen::Vector3i operator() (const Voxel& voxel) const {
        return voxel.grid_index_;
    }
};

struct add_voxel_color_functor {
    __device__
    Voxel operator() (const Voxel& x, const Voxel& y) const {
        Voxel ans;
        ans.grid_index_ = x.grid_index_;
        ans.color_ = x.color_ + y.color_;
        return ans;
    }
};

struct devide_voxel_color_functor {
    __device__
    Voxel operator() (const Voxel& x, int y) const {
        Voxel ans;
        ans.grid_index_ = x.grid_index_;
        ans.color_ = x.color_ / y;
        return ans;
    }
};

__host__ __device__
void GetVoxelBoundingPoints(const Eigen::Vector3f& x, float r,
                            Eigen::Vector3f points[8]) {
    points[0] = x + Eigen::Vector3f(-r, -r, -r);
    points[1] = x + Eigen::Vector3f(-r, -r, r);
    points[2] = x + Eigen::Vector3f(r, -r, -r);
    points[3] = x + Eigen::Vector3f(r, -r, r);
    points[4] = x + Eigen::Vector3f(-r, r, -r);
    points[5] = x + Eigen::Vector3f(-r, r, r);
    points[6] = x + Eigen::Vector3f(r, r, -r);
    points[7] = x + Eigen::Vector3f(r, r, r);
}

struct compute_carve_functor {
    compute_carve_functor(const uint8_t* image, int width, int height,
                          int num_of_channels, int bytes_per_channel,
                          float voxel_size, const Eigen::Vector3f& origin,
                          const Eigen::Matrix3f& intrinsic,
                          const Eigen::Matrix3f& rot, const Eigen::Vector3f& trans,
                          bool keep_voxels_outside_image)
                          : image_(image), width_(width), height_(height),
                            num_of_channels_(num_of_channels), bytes_per_channel_(bytes_per_channel),
                            voxel_size_(voxel_size), origin_(origin),
                            intrinsic_(intrinsic), rot_(rot), trans_(trans),
                            keep_voxels_outside_image_(keep_voxels_outside_image) {};
    const uint8_t* image_;
    const int width_;
    const int height_;
    const int num_of_channels_;
    const int bytes_per_channel_;
    const float voxel_size_;
    const Eigen::Vector3f origin_;
    const Eigen::Matrix3f intrinsic_;
    const Eigen::Matrix3f rot_;
    const Eigen::Vector3f trans_;
    bool keep_voxels_outside_image_;
    __device__
    bool operator() (const thrust::tuple<Eigen::Vector3i, Voxel>& voxel) const {
        bool carve = true;
        float r = voxel_size_ / 2.0;
        Voxel v = thrust::get<1>(voxel);
        auto x = ((v.grid_index_.cast<float>() + Eigen::Vector3f(0.5, 0.5, 0.5)) * voxel_size_) + origin_;
        Eigen::Vector3f pts[8];
        GetVoxelBoundingPoints(x, r, pts);
        for (int i = 0; i < 8; ++i) {
            auto x_trans = rot_ * pts[i] + trans_;
            auto uvz = intrinsic_ * x_trans;
            float z = uvz(2);
            float u = uvz(0) / z;
            float v = uvz(1) / z;
            float d;
            bool within_boundary;
            thrust::tie(within_boundary, d) = FloatValueAt(image_,
                                                           u, v, width_, height_,
                                                           num_of_channels_, bytes_per_channel_);
            if ((!within_boundary && keep_voxels_outside_image_) ||
                (within_boundary && d > 0 && z >= d)) {
                carve = false;
                break;
            }
        }
        return carve;
    }
};

}

VoxelGrid::VoxelGrid() : Geometry3D(Geometry::GeometryType::VoxelGrid) {}
VoxelGrid::~VoxelGrid() {}

VoxelGrid::VoxelGrid(const VoxelGrid &src_voxel_grid)
    : Geometry3D(Geometry::GeometryType::VoxelGrid),
      voxel_size_(src_voxel_grid.voxel_size_),
      origin_(src_voxel_grid.origin_),
      voxels_keys_(src_voxel_grid.voxels_keys_),
      voxels_values_(src_voxel_grid.voxels_values_) {}

VoxelGrid &VoxelGrid::Clear() {
    voxel_size_ = 0.0;
    origin_ = Eigen::Vector3f::Zero();
    voxels_keys_.clear();
    voxels_values_.clear();
    return *this;
}

bool VoxelGrid::IsEmpty() const { return !HasVoxels(); }

Eigen::Vector3f VoxelGrid::GetMinBound() const {
    if (!HasVoxels()) {
        return origin_;
    } else {
        Voxel v = voxels_values_[0];
        Eigen::Vector3i init = v.grid_index_;
        Eigen::Vector3i min_grid_index = thrust::reduce(thrust::make_transform_iterator(voxels_values_.begin(), extract_grid_index_functor()),
                                                        thrust::make_transform_iterator(voxels_values_.end(), extract_grid_index_functor()),
                                                        init, elementwise_min_functor());
        return min_grid_index.cast<float>() * voxel_size_ + origin_;
    }
}

Eigen::Vector3f VoxelGrid::GetMaxBound() const {
    if (!HasVoxels()) {
        return origin_;
    } else {
        Voxel v = voxels_values_[0];
        Eigen::Vector3i init = v.grid_index_;
        Eigen::Vector3i min_grid_index = thrust::reduce(thrust::make_transform_iterator(voxels_values_.begin(), extract_grid_index_functor()),
                                                        thrust::make_transform_iterator(voxels_values_.end(), extract_grid_index_functor()),
                                                        init, elementwise_max_functor());
        return (min_grid_index.cast<float>() + Eigen::Vector3f::Ones()) * voxel_size_ + origin_;
    }
}

Eigen::Vector3f VoxelGrid::GetCenter() const {
    Eigen::Vector3f init(0, 0, 0);
    if (!HasVoxels()) {
        return init;
    }
    const Eigen::Vector3f half_voxel_size(0.5 * voxel_size_, 0.5 * voxel_size_,
                                          0.5 * voxel_size_);
    compute_center_functor func(voxel_size_, origin_, half_voxel_size);
    Eigen::Vector3f center = thrust::transform_reduce(thrust::make_transform_iterator(voxels_values_.begin(), extract_grid_index_functor()),
                                                      thrust::make_transform_iterator(voxels_values_.end(), extract_grid_index_functor()),
                                                      func, init, thrust::plus<Eigen::Vector3f>());
    center /= float(voxels_values_.size());
    return center;
}

AxisAlignedBoundingBox VoxelGrid::GetAxisAlignedBoundingBox() const {
    AxisAlignedBoundingBox box;
    box.min_bound_ = GetMinBound();
    box.max_bound_ = GetMaxBound();
    return box;
}

OrientedBoundingBox VoxelGrid::GetOrientedBoundingBox() const {
    return OrientedBoundingBox::CreateFromAxisAlignedBoundingBox(
            GetAxisAlignedBoundingBox());
}

VoxelGrid &VoxelGrid::Transform(const Eigen::Matrix4f &transformation) {
    utility::LogError("VoxelGrid::Transform is not supported");
    return *this;
}

VoxelGrid &VoxelGrid::Translate(const Eigen::Vector3f &translation,
                                bool relative) {
    utility::LogError("Not implemented");
    return *this;
}

VoxelGrid &VoxelGrid::Scale(const float scale, bool center) {
    utility::LogError("Not implemented");
    return *this;
}

VoxelGrid &VoxelGrid::Rotate(const Eigen::Matrix3f &R, bool center) {
    utility::LogError("VoxelGrid::Transform is not supported");
    return *this;
}

VoxelGrid &VoxelGrid::operator+=(const VoxelGrid &voxelgrid) {
    if (voxel_size_ != voxelgrid.voxel_size_) {
        utility::LogError(
                "[VoxelGrid] Could not combine VoxelGrid because voxel_size "
                "differs (this=%f, other=%f)",
                voxel_size_, voxelgrid.voxel_size_);
    }
    if (origin_ != voxelgrid.origin_) {
        utility::LogError(
                "[VoxelGrid] Could not combine VoxelGrid because origin "
                "differs (this=%f,%f,%f, other=%f,%f,%f)",
                origin_(0), origin_(1), origin_(2), voxelgrid.origin_(0),
                voxelgrid.origin_(1), voxelgrid.origin_(2));
    }
    if (this->HasColors() != voxelgrid.HasColors()) {
        utility::LogError(
                "[VoxelGrid] Could not combine VoxelGrid one has colors and "
                "the other not.");
    }
    if (voxelgrid.HasColors()) {
        voxels_keys_.insert(voxels_keys_.end(), voxelgrid.voxels_keys_.begin(), voxelgrid.voxels_keys_.end());
        voxels_values_.insert(voxels_values_.end(), voxelgrid.voxels_values_.begin(), voxelgrid.voxels_values_.end());
        thrust::sort_by_key(voxels_keys_.begin(), voxels_keys_.end(), voxels_values_.begin());
        thrust::device_vector<int> counts(voxels_keys_.size());
        thrust::device_vector<Eigen::Vector3i> new_keys(voxels_keys_.size());
        auto end1 = thrust::reduce_by_key(voxels_keys_.begin(), voxels_keys_.end(),
                                          thrust::make_constant_iterator(1),
                                          thrust::make_discard_iterator(), counts.begin());
        int n_out = thrust::distance(counts.begin(), end1.second);
        counts.resize(n_out);
        auto end2 = thrust::reduce_by_key(voxels_keys_.begin(), voxels_keys_.end(),
                                          voxels_values_.begin(), new_keys.begin(),
                                          voxels_values_.begin(),
                                          thrust::equal_to<Eigen::Vector3i>(),
                                          add_voxel_color_functor());
        new_keys.resize(n_out);
        voxels_keys_ = new_keys;
        voxels_values_.resize(n_out);
        thrust::transform(voxels_values_.begin(), voxels_values_.end(),
                          counts.begin(), voxels_values_.begin(),
                          devide_voxel_color_functor());
    } else {
        this->AddVoxels(voxelgrid.voxels_values_);
    }
    return *this;
}

VoxelGrid VoxelGrid::operator+(const VoxelGrid &voxelgrid) const {
    return (VoxelGrid(*this) += voxelgrid);
}

void VoxelGrid::AddVoxel(const Voxel &voxel) {
    voxels_keys_.push_back(voxel.grid_index_);
    voxels_values_.push_back(voxel);
    thrust::sort_by_key(voxels_keys_.begin(), voxels_keys_.end(), voxels_values_.begin());
    auto end = thrust::unique_by_key(voxels_keys_.begin(), voxels_keys_.end(), voxels_values_.begin());
    size_t out_size = thrust::distance(voxels_keys_.begin(), end.first);
    voxels_keys_.resize(out_size);
    voxels_values_.resize(out_size);
}

void VoxelGrid::AddVoxels(const thrust::device_vector<Voxel> &voxels) {
    voxels_keys_.insert(voxels_keys_.end(),
                        thrust::make_transform_iterator(voxels.begin(), extract_grid_index_functor()),
                        thrust::make_transform_iterator(voxels.end(), extract_grid_index_functor()));
    voxels_values_.insert(voxels_values_.end(), voxels.begin(), voxels.end());
    thrust::sort_by_key(voxels_keys_.begin(), voxels_keys_.end(), voxels_values_.begin());
    auto end = thrust::unique_by_key(voxels_keys_.begin(), voxels_keys_.end(), voxels_values_.begin());
    size_t out_size = thrust::distance(voxels_keys_.begin(), end.first);
    voxels_keys_.resize(out_size);
    voxels_values_.resize(out_size);
}

Eigen::Vector3i VoxelGrid::GetVoxel(const Eigen::Vector3f &point) const {
    Eigen::Vector3f voxel_f = (point - origin_) / voxel_size_;
    return (Eigen::floor(voxel_f.array())).cast<int>();
}

Eigen::Vector3f VoxelGrid::GetVoxelCenterCoordinate(const Eigen::Vector3i &idx) const {
    auto it = thrust::find(voxels_keys_.begin(), voxels_keys_.end(), idx);
    if (it != voxels_keys_.end()) {
        Eigen::Vector3i voxel_idx = *it;
        return ((voxel_idx.cast<float>() +
                 Eigen::Vector3f(0.5, 0.5, 0.5)) *
                voxel_size_) +
               origin_;
    } else {
        return Eigen::Vector3f::Zero();
    }
}

std::array<Eigen::Vector3f, 8> VoxelGrid::GetVoxelBoundingPoints(
        const Eigen::Vector3i &index) const {
    float r = voxel_size_ / 2.0;
    auto x = GetVoxelCenterCoordinate(index);
    std::array<Eigen::Vector3f, 8> points;
    ::GetVoxelBoundingPoints(x, r, points.data());
    return points;
}

thrust::host_vector<bool> VoxelGrid::CheckIfIncluded(
        const thrust::host_vector<Eigen::Vector3f> &queries) {
    thrust::host_vector<bool> output;
    output.resize(queries.size());
    for (size_t i = 0; i < queries.size(); ++i) {
        auto query = GetVoxel(queries[i]);
        auto itr = thrust::find(thrust::cuda::par.on(utility::GetStream(i % utility::MAX_NUM_STREAMS)),
                                voxels_keys_.begin(), voxels_keys_.end(), query);
        output[i] = (itr != voxels_keys_.end());
    }
    cudaSafeCall(hipDeviceSynchronize());
    return output;
}

VoxelGrid &VoxelGrid::CarveDepthMap(
        const Image &depth_map,
        const camera::PinholeCameraParameters &camera_parameter,
        bool keep_voxels_outside_image) {
    if (depth_map.height_ != camera_parameter.intrinsic_.height_ ||
        depth_map.width_ != camera_parameter.intrinsic_.width_) {
        utility::LogError(
                "[VoxelGrid] provided depth_map dimensions are not compatible "
                "with the provided camera_parameters");
    }

    auto rot = camera_parameter.extrinsic_.block<3, 3>(0, 0);
    auto trans = camera_parameter.extrinsic_.block<3, 1>(0, 3);
    auto intrinsic = camera_parameter.intrinsic_.intrinsic_matrix_;

    // get for each voxel if it projects to a valid pixel and check if the voxel
    // depth is behind the depth of the depth map at the projected pixel.
    compute_carve_functor func(thrust::raw_pointer_cast(depth_map.data_.data()),
                               depth_map.width_, depth_map.height_,
                               depth_map.num_of_channels_, depth_map.bytes_per_channel_,
                               voxel_size_, origin_,
                               intrinsic, rot, trans, keep_voxels_outside_image);
    auto begin = make_tuple_iterator(voxels_keys_.begin(), voxels_values_.begin());
    auto end = thrust::remove_if(begin, make_tuple_iterator(voxels_keys_.end(), voxels_values_.end()), func);
    size_t out_size = thrust::distance(begin, end);
    voxels_keys_.resize(out_size);
    voxels_values_.resize(out_size);
    return *this;
}

VoxelGrid &VoxelGrid::CarveSilhouette(
        const Image &silhouette_mask,
        const camera::PinholeCameraParameters &camera_parameter,
        bool keep_voxels_outside_image) {
    if (silhouette_mask.height_ != camera_parameter.intrinsic_.height_ ||
        silhouette_mask.width_ != camera_parameter.intrinsic_.width_) {
        utility::LogError(
                "[VoxelGrid] provided silhouette_mask dimensions are not "
                "compatible with the provided camera_parameters");
    }

    auto rot = camera_parameter.extrinsic_.block<3, 3>(0, 0);
    auto trans = camera_parameter.extrinsic_.block<3, 1>(0, 3);
    auto intrinsic = camera_parameter.intrinsic_.intrinsic_matrix_;

    // get for each voxel if it projects to a valid pixel and check if the pixel
    // is set (>0).
    compute_carve_functor func(thrust::raw_pointer_cast(silhouette_mask.data_.data()),
                               silhouette_mask.width_, silhouette_mask.height_,
                               silhouette_mask.num_of_channels_, silhouette_mask.bytes_per_channel_,
                               voxel_size_, origin_,
                               intrinsic, rot, trans, keep_voxels_outside_image);
    auto begin = make_tuple_iterator(voxels_keys_.begin(), voxels_values_.begin());
    auto end = thrust::remove_if(begin, make_tuple_iterator(voxels_keys_.end(), voxels_values_.end()), func);
    size_t out_size = thrust::distance(begin, end);
    voxels_keys_.resize(out_size);
    voxels_values_.resize(out_size);
    return *this;
}
